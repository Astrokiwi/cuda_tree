
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <random>
#include <fstream>

__device__ float square(float x) {
    return x*x;
}

struct Tree_Info {
    int n;
    double corner[3];
    double size; // assume cube
};

// Just build the tree in serial
// DOES NOT WORK
void buildtree(int n, float *r, float *h, float *coldens, int *tree_children, Tree_Info *tree_info) {
    // find max/min size
    int ii,kk;
    float *r_i;

    for ( kk=0 ; kk<3 ; kk++) tree_info->corner[kk] = r[kk];
    tree_info->size = 0.;

    for ( ii=1 ; ii<n ; ii++ ) {
        r_i = &(r[ii*3]);
        
        for ( kk=0 ; kk<3 ; kk++ ) {
            if ( r_i[kk]<tree_info->corner[kk] ) {
                tree_info->size+=tree_info->corner[kk]-r_i[kk];
                tree_info->corner[kk]-=r_i[kk];
            }
            if ( r_i[kk]>tree_info->corner[kk] ) {
                tree_info->size+=r_i[kk]-tree_info->corner[kk];
            }
        }
    }
    
    std::cout << "Corner, size: " << tree_info->corner[0] << " " << tree_info->corner[1] << " " << tree_info->corner[2] << " " << tree_info->size << std::endl;
}

// CUDA Kernel function function to calc number of collisions per particle
__global__ void calcray(int n, float *r, float *h, float *coldens, Tree_Info *tree_info) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = blockDim.x * gridDim.x;
  
  int i,j,k;

  float crossp;
  
  float norm2,h2,dot;
  float *r_i,*r_j;
  float dr[3];

  i = index;
  coldens[i] = 0.;

  r_i = &(r[i*3]);
  norm2 = 0.;
  for (k=0 ; k<3 ; k++) norm2+=square(r_i[k]);

  for (j=0 ; j<n ; j++) {
    r_j = &(r[j*3]);
    // check if particle is in-between origin and target
    dot = 0.;
    for (k=0 ; k<3 ; k++) dot+=r_i[k]*r_j[k];

    if ( dot>0. && dot<norm2 ) {
        // check if ray intersects particle
        for (k=0 ; k<3 ; k++) dr[k] = r_i[k] - r_j[k];

        crossp = square( r_j[1]*dr[2]-r_j[2]*dr[1]);
        crossp+= square(-r_j[0]*dr[2]+r_j[2]*dr[0]);
        crossp+= square( r_j[0]*dr[1]-r_j[1]*dr[0]);
  
        h2 = square(h[j]);

        if ( crossp<=h2*norm2 ) {
          coldens[index]+=1.;
        }
    }
  }
}

int main(void) {
//   int N = 690286; // for comparison with tree code
  int N = 10000; // for quick tests

  float *r,*h,*coldens;
  float *d_coldens;
  
  int *tree_children;
  
  struct Tree_Info *tree_info;
  
  int tree_memory_factor = 8;

  hipMallocManaged(&r, N*sizeof(float)*3);
  hipMallocManaged(&h, N*sizeof(float));
  hipMalloc(&d_coldens, N*sizeof(float));

  hipMallocManaged(&tree_children, N*sizeof(int)*tree_memory_factor);
  hipMallocManaged(&tree_info, sizeof(tree_info));

  coldens = new float[N];
  
  std::default_random_engine generator;
  std::uniform_real_distribution<float> locDistribution(-1.0,1.0);
//   std::uniform_real_distribution<float> hDistribution(0.005,0.02);
  std::uniform_real_distribution<float> hDistribution(0.05,0.1);

  
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    r[i*3] = locDistribution(generator);
    r[i*3+1] = locDistribution(generator);
    r[i*3+2] = locDistribution(generator);
    h[i] = hDistribution(generator);
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  buildtree(N,r,h,d_coldens,tree_children,tree_info);

  calcray<<<numBlocks, blockSize>>>(N,r,h,d_coldens,tree_info);
  hipMemcpy(coldens, d_coldens, N*sizeof(float), hipMemcpyDeviceToHost);
// 
  std::cout << "coldens0  " << coldens[0] << std::endl;
  
  // Free memory
  hipFree(r);
  hipFree(h);
  hipFree(coldens);
  
  hipFree(d_coldens);
  hipFree(tree_children);
  hipFree(tree_info);

  return 0;
}
